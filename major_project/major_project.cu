#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <omp.h>

using namespace std;

// CUDA kernel for matrix addition
__global__ void matrixAddKernel(int n, int* A, int* B, int* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        C[row * n + col] = A[row * n + col] + B[row * n + col];
    }
}

// CUDA kernel for matrix subtraction
__global__ void matrixSubKernel(int n, int* A, int* B, int* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        C[row * n + col] = A[row * n + col] - B[row * n + col];
    }
}

// CUDA kernel for standard matrix multiplication
__global__ void matrixMulKernel(int n, int* A, int* B, int* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

// Allocate and initialize matrix on host
void matrixRand(int n, int*& matrix) {
    matrix = (int*)malloc(n * n * sizeof(int));
    for (int i = 0; i < n * n; i++) {
        matrix[i] = rand() % 1000;
    }
}

// Print matrix
void matrixPrint(int n, int* matrix) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            cout << matrix[i * n + j] << " ";
        }
        cout << endl;
    }
}

int** matrixAllocate(int n) {
    int **matrix = new int*[n];
    for (int i = 0; i < n; i++) {
        matrix[i] = new int[n];
    }
    return matrix;
}

void matrixFree(int **matrix, int n) {
    for (int i = 0; i < n; i++) {
        delete [] matrix[i];
    }
    delete [] matrix;
}

bool matrixCompare(int n, int** A, int** B) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (A[i][j] != B[i][j]) return false;
        }
    }
    return true;
}

void matrixStandardMul(int n, int** A, int** B, int** C) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            C[i][j] = 0;
            for (int k = 0; k < n; k++) {
                C[i][j] += A[i][k] * B[k][j];
            }
        }
    }
}

// Main function
int main(int argc, char* argv[]) {
    struct timespec start, stop, start_standard, stop_standard;
    double total_time, total_time_standard;

    if (argc != 3) {
        printf("Need two integers as input \n");
        printf("Use: <executable_name> <log_2(list_size)> <log_2(terminal_size)>\n");
        exit(0);
    }

    // k and k'
    int n = (1 << atoi(argv[1]));
    int threshold = (1 << atoi(argv[2]));

    // Allocate new matrices
    int** A = matrixAllocate(n);
    int** B = matrixAllocate(n);
    int** C = matrixAllocate(n);
    int** Cseq = matrixAllocate(n);

    // Randomly generate matrix values
    matrixRand(n, *A);
    matrixRand(n, *B);

    // Strassen Multiplication using OpenMP
    clock_gettime(CLOCK_REALTIME, &start);
    omp_set_num_threads(8);
    #pragma omp parallel
    {
        #pragma omp single
        {
            // Placeholder function for Strassen's Algorithm
            if (n <= threshold) {
                matrixStandardMul(n, A, B, C); // Use standard multiplication for small matrices
            } else {
                // Implement Strassen's Algorithm here if available
                matrixStandardMul(n, A, B, C); // Currently using standard multiplication as a placeholder
            }
        }
    }
    clock_gettime(CLOCK_REALTIME, &stop);
    total_time = (stop.tv_sec - start.tv_sec) + 0.000000001 * (stop.tv_nsec - start.tv_nsec);

    // Standard Multiplication
    clock_gettime(CLOCK_REALTIME, &start_standard);
    matrixStandardMul(n, A, B, Cseq);
    clock_gettime(CLOCK_REALTIME, &stop_standard);
    total_time_standard = (stop_standard.tv_sec - start_standard.tv_sec) + 0.000000001 * (stop_standard.tv_nsec - start_standard.tv_nsec);

    // Check answer
    if (matrixCompare(n, C, Cseq)) {
        cout << "Correct!!!" << endl;
        printf("Matrix Size = %d * %d, Threshold = %d, time (sec) = %8.4f, standard_time = %8.4f\n", 
               n, n, threshold, total_time, total_time_standard);
    } else {
        cout << "We have a problem!" << endl;
    }

    // Free allocated memory
    matrixFree(A, n);
    matrixFree(B, n);
    matrixFree(C, n);
    matrixFree(Cseq, n);

    return 0;
}
